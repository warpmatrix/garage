#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() { printf("Hello Cuda!\n"); }

int main(int argc, char const *argv[]) {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
