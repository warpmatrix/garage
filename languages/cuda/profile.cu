#include <cstdio>
#include <functional>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nvtx3/nvtx3.hpp>

class CudaProfiler{
public:
    CudaProfiler() {
        hipProfilerStart();
    }
    ~CudaProfiler() {
        hipProfilerStop();
    }
};

void with_cuda_profiler(std::function<void ()> fn) {
    CudaProfiler p;
    fn();
}

__global__ void longKernel() {
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    clock_t clock_count = 1000000000;
    while (clock_offset < clock_count) {
        clock_offset = clock() - start_clock;
    }
}

void launch_long_kernel() {
    NVTX3_FUNC_RANGE();
    longKernel<<<1, 1>>>();
}

int main() {
    with_cuda_profiler([]() {
        NVTX3_FUNC_RANGE();
        launch_long_kernel();
        hipDeviceSynchronize();
    });
    return 0;
}
