#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void wait() {
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    clock_t clock_count = 10000000000;
    while (clock_offset < clock_count) {
        clock_offset = clock() - start_clock;
    }
}

int main(int argc, char const *argv[]) {
    wait<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
