#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(int *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] *= 2;
}

int main() {
    const int size = 1024;
    int *d_a, *d_b;
    int h_a[size], h_b[size];

    // 分配设备内存
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));

    // 初始化数据
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i + 1;
    }

    // 将数据从主机拷贝到设备
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // 创建 CUDA 流
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // 在流 1 中执行 kernel
    kernel<<<1, 256, 0, stream1>>>(d_a);

    // 在流 2 中执行 D2D memcpy
    hipMemcpyAsync(d_b, d_a, size * sizeof(int), hipMemcpyDeviceToDevice, stream2);

    // 等待流中的操作完成
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // 释放设备内存
    hipFree(d_a);
    hipFree(d_b);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
