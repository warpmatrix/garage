#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void wait() {
    clock_t start = clock();
    clock_t clock_offset = 0;
    clock_t clock_count = 100000000;
    while (clock_offset < clock_count) {
        clock_offset = clock() - start;
    }
}

int main(int argc, char const *argv[]) {
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    wait<<<1, 1>>>();
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);
    printf("elapsed time: %f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(end);
    return 0;
}
