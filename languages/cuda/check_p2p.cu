#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    for (int i = 0; i < deviceCount; ++i) {
        for (int j = 0; j < deviceCount; ++j) {
            if (i == j) {
                continue;
            }
            int canAccessPeer;
            hipDeviceCanAccessPeer(&canAccessPeer, i, j);
            std::cout << "Device " << i << " can ";
            if (!canAccessPeer) std::cout << "not ";
            std::cout << "access Device " << j << " via P2P" << std::endl;
        }
    }

    return 0;
}
