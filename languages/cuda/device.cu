#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {
    int driver_version = 0, runtime_version = 0;

    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("Driver Version: %d, Runtime Version: %d\n",
        driver_version, runtime_version);

    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (size_t i = 0; i < nDevices; i++) {
        hipDeviceProp_t p;
        hipGetDeviceProperties(&p, i);
        printf(" Device Number: %lu\n", i);
        printf(" Device name: %s\n", p.name);
        printf(" Memory Clock Rate (KHz): %d\n", p.memoryClockRate);
        printf(" Memory Bus Width (bits): %d\n", p.memoryBusWidth);
        printf(" Device %d has compute capability %d.%d.\n",
            device, deviceProp.major, deviceProp.minor);
        printf(" Device %d support: concurrent kenel execution(%d), intra-device copy(%d)\n",
            device, deviceProp.concurrentKernels, deviceProp.asyncEngineCount);
    }

    return 0;
}
